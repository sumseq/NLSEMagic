#include "hip/hip_runtime.h"

/*Define block size.*/
const int BLOCK_SIZE = 512;

/*Kernel to evaluate F(Psi) using shared memory*/
__global__ void compute_F_2SHOC_F(float* ktotr, float* ktoti,
                                  float* Utmpr, float* Utmpi,
                                  float* Uoldr, float* Uoldi,
                                  float* Uoutr, float* Uouti,
                                  float* Dr,    float* Di,
                                  float* V, float s, float a,float a76,float a112,
                                  int BC, int N, float k, int fstep)
{
    /*Declare shared memory space*/
    __shared__ float sUtmpr[BLOCK_SIZE+2];
    __shared__ float sUtmpi[BLOCK_SIZE+2];
    __shared__ float  NLSFr[BLOCK_SIZE+2];
    __shared__ float  NLSFi[BLOCK_SIZE+2];
    __shared__ float    sDr[BLOCK_SIZE+2];
    __shared__ float    sDi[BLOCK_SIZE+2];
    __shared__ float     sV[BLOCK_SIZE+2];

    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x+threadIdx.x;
    int si = threadIdx.x+1;
    float OM;
    int msd_si;

    /*Copy vectors from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
        sDr[si]    = Dr[i];
        sDi[si]    = Di[i];
        sV[si]     = V[i];
    }

    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i >0 && i< N-1)
    {
        if(si==1)
        {
            sDr[0] =  Dr[i-1];
            sDi[0] =  Di[i-1];
        }
        if(si==blockDim.x)
        {
           sDr[si+1] = Dr[i+1];
           sDi[si+1] = Di[i+1];
        }
        /*No synchthreads needed in this case*/
        NLSFr[si] = a112*(sDi[si+1] + sDi[si-1]) - a76*sDi[si]
                 - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
        NLSFi[si] =  a76*sDr[si] - a112*(sDr[si+1] + sDr[si-1])
                 + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
    }/*End of interier points*/

    if(BC==2)   __syncthreads(); /* needed for MSD*/

    /*Boundery Conditions*/
    if(i == 0 || i == N-1){

        /*Boundary conditions:*/
        switch (BC){
            case 1:
                NLSFr[si]   = 0.0f;
                NLSFi[si]   = 0.0f;
                break;
            case 2:
                if(i==0)   msd_si = si+1;
                if(i==N-1) msd_si = si-1;

                OM = (NLSFi[msd_si]*sUtmpr[msd_si]  - NLSFr[msd_si]*sUtmpi[msd_si])/
                     (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);
                                        
                NLSFr[si]  = -OM*sUtmpi[si];
                NLSFi[si]  =  OM*sUtmpr[si];
                break;
            case 3:
                NLSFr[si]   = - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si]   =   (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            case 4:
                NLSFr[si] = -a*sDi[si] - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si] =  a*sDr[si] + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            default:
                NLSFr[si]   = 0.0f;
                NLSFi[si]   = 0.0f;
                break;
       }/*BC Switch*/
    }/*BC*/

    if(i<N){
      switch(fstep)  {
          case 1:
            ktotr[i] = NLSFr[si];
            ktoti[i] = NLSFi[si];
            /*sUtmp is really Uold and Uold is really Utmp*/
            Uoldr[i] = sUtmpr[si] + k*NLSFr[si];
            Uoldi[i] = sUtmpi[si] + k*NLSFi[si];
            break;
          case 2:
            ktotr[i] = ktotr[i] + 2*NLSFr[si];
            ktoti[i] = ktoti[i] + 2*NLSFi[si];
            Uoutr[i] = Uoldr[i] + k*NLSFr[si];
            Uouti[i] = Uoldi[i] + k*NLSFi[si];
            break;
          case 3:
            Uoldr[i] = Uoldr[i] + k*(ktotr[i] + NLSFr[si]);
            Uoldi[i] = Uoldi[i] + k*(ktoti[i] + NLSFi[si]);
            break;
        }/*switch step*/
    }/*i<N*/

}/*Compute_F*/

/*Kernel to evaluate D(Psi) using shared memory*/
__global__ void compute_D_F  (float* Dr,    float* Di,
                              float* Utmpr, float* Utmpi,
                              float* V,
                              float lh2,  float l_a, float s,int BC, int N)
{
    /*Declare shared memory space*/
    __shared__ float sUtmpr[BLOCK_SIZE+2];
    __shared__ float sUtmpi[BLOCK_SIZE+2];
    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x+threadIdx.x;
    int si = threadIdx.x+1;
    float A,Nb,Nb1;
    int msd_si,msd_i;

    /*Copy vector from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
    }
    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i >0 && i < N-1)
    {
        if(si==1)
        {
            sUtmpr[0] = Utmpr[i-1];
            sUtmpi[0] = Utmpi[i-1];
        }
            if(si==blockDim.x)
            {
                sUtmpr[si+1] = Utmpr[i+1];
                sUtmpi[si+1] = Utmpi[i+1];
            }
        /*No synchthreads needed in this case*/

        Dr[i] = (sUtmpr[si+1] - 2*sUtmpr[si] + sUtmpr[si-1])*lh2;
        Di[i] = (sUtmpi[si+1] - 2*sUtmpi[si] + sUtmpi[si-1])*lh2;
    }

    if(BC==2)   __syncthreads(); /* needed for MSD*/

        /*Boundery Conditions*/
    if(i == 0 || i == N-1){

        /*Boundary conditions:*/
        switch (BC){
            case 1: /*Dirichlet*/
                Dr[i] = -l_a*(s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - V[i])*sUtmpr[si];
                Di[i] = -l_a*(s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - V[i])*sUtmpi[si];
                break;
            case 2:
                if(i==0)                msd_si = si+1;
                if(i==N-1)              msd_si = si-1;
                msd_i = i+(msd_si-si);

                Nb  = s*(sUtmpr[si]*sUtmpr[si]         + sUtmpi[si]*sUtmpi[si])         - V[i];
                Nb1 = s*(sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]) - V[msd_i];

                A   = (Dr[msd_i]*sUtmpr[msd_si]      + Di[msd_i]*sUtmpi[msd_si])/
                      (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);
                    
                Dr[i]  = (A + l_a*(Nb1-Nb))*sUtmpr[si];
                Di[i]  = (A + l_a*(Nb1-Nb))*sUtmpi[si];
                break;
            case 3:
                Dr[i]   = 0.0f;
                Di[i]   = 0.0f;
                break;
            case 4:
                if(i==0){
                   Dr[i] = (-Utmpr[3] + 4*Utmpr[2] - 5*sUtmpr[si+1] + 2*sUtmpr[si])*lh2;
                   Di[i] = (-Utmpi[3] + 4*Utmpi[2] - 5*sUtmpi[si+1] + 2*sUtmpi[si])*lh2;
                }
                else{
                   Dr[i] = (-Utmpr[N-4] + 4*Utmpr[N-3] - 5*sUtmpr[si-1] + 2*sUtmpr[si])*lh2;
                   Di[i] = (-Utmpi[N-4] + 4*Utmpi[N-3] - 5*sUtmpi[si-1] + 2*sUtmpi[si])*lh2;
                }
                break;
            default:
                Dr[i]   = 0.0f;
                Di[i]   = 0.0f;
                break;
       }/*BC Switch*/
    }/*BC*/
}/*computedx2*/


/*Kernel to evaluate F(Psi) using shared memory*/
__global__ void compute_F_2SHOC(double* ktotr, double* ktoti,
                                double* Utmpr, double* Utmpi,
                                double* Uoldr, double* Uoldi,
                                double* Uoutr, double* Uouti,
                                double* Dr,    double* Di,
                                double* V, double s, double a,double a76,double a112,
                                int BC, int N, double k, int fstep)
{
    /*Declare shared memory space*/
    __shared__ double sUtmpr[BLOCK_SIZE+2];
    __shared__ double sUtmpi[BLOCK_SIZE+2];
    __shared__ double  NLSFr[BLOCK_SIZE+2];
    __shared__ double  NLSFi[BLOCK_SIZE+2];
    __shared__ double    sDr[BLOCK_SIZE+2];
    __shared__ double    sDi[BLOCK_SIZE+2];
    __shared__ double     sV[BLOCK_SIZE+2];

    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x+threadIdx.x;
    int si = threadIdx.x+1;
    double OM;
    int msd_si;

    /*Copy vectors from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
        sDr[si]    = Dr[i];
        sDi[si]    = Di[i];
        sV[si]     = V[i];
    }

    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i >0 && i< N-1)
    {
        if(si==1)
        {
           sDr[0] =  Dr[i-1];
           sDi[0] =  Di[i-1];
        }
        if(si==blockDim.x)
        {
           sDr[si+1] = Dr[i+1];
           sDi[si+1] = Di[i+1];
        }
        /*No synchthreads needed in this case*/
        NLSFr[si] = a112*(sDi[si+1] + sDi[si-1]) - a76*sDi[si]
                 - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
        NLSFi[si] =  a76*sDr[si] - a112*(sDr[si+1] + sDr[si-1])
                 + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
    }/*End of interier points*/

    if(BC==2)   __syncthreads(); /* needed for MSD*/

    /*Boundery Conditions*/
    if(i == 0 || i == N-1){

        /*Boundary conditions:*/
        switch (BC){
            case 1:
                NLSFr[si]   = 0.0;
                NLSFi[si]   = 0.0;
                break;
            case 2:
                if(i==0)     msd_si = si+1;
                if(i==N-1)   msd_si = si-1;

                OM = (NLSFi[msd_si]*sUtmpr[msd_si]  - NLSFr[msd_si]*sUtmpi[msd_si])/
                     (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);
                                        
                NLSFr[si]  = -OM*sUtmpi[si];
                NLSFi[si]  =  OM*sUtmpr[si];
                break;
            case 3:
                NLSFr[si]   = - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si]   =   (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            case 4:
                NLSFr[si] = -a*sDi[si] - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si] =  a*sDr[si] + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            default:
                NLSFr[si]   = 0.0;
                NLSFi[si]   = 0.0;
                break;
       }/*BC Switch*/
    }/*BC*/

    if(i<N){
      switch(fstep)  {
          case 1:
            ktotr[i] = NLSFr[si];
            ktoti[i] = NLSFi[si];
            /*sUtmp is really Uold and Uold is really Utmp*/
            Uoldr[i] = sUtmpr[si] + k*NLSFr[si];
            Uoldi[i] = sUtmpi[si] + k*NLSFi[si];
            break;
          case 2:
            ktotr[i] = ktotr[i] + 2*NLSFr[si];
            ktoti[i] = ktoti[i] + 2*NLSFi[si];
            Uoutr[i] = Uoldr[i] + k*NLSFr[si];
            Uouti[i] = Uoldi[i] + k*NLSFi[si];
            break;
          case 3:
            Uoldr[i] = Uoldr[i] + k*(ktotr[i] + NLSFr[si]);
            Uoldi[i] = Uoldi[i] + k*(ktoti[i] + NLSFi[si]);
            break;
        }/*switch step*/
    }/*i<N*/

}/*Compute_F_2SHOC*/

/*Kernel to evaluate D(Psi) using shared memory*/
__global__ void compute_D  (double* Dr,    double* Di,
                            double* Utmpr, double* Utmpi,
                            double* V,
                            double lh2,  double l_a, double s,int BC, int N)
{
    /*Declare shared memory space*/
    __shared__ double sUtmpr[BLOCK_SIZE+2];
    __shared__ double sUtmpi[BLOCK_SIZE+2];
    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x+threadIdx.x;
    int si = threadIdx.x+1;
    double A,Nb,Nb1;
    int msd_si,msd_i;

    /*Copy vector from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
    }
    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i >0 && i < N-1)
    {
        if(si==1)
        {
            sUtmpr[0] = Utmpr[i-1];
            sUtmpi[0] = Utmpi[i-1];
        }
        if(si==blockDim.x)
        {
            sUtmpr[si+1] = Utmpr[i+1];
            sUtmpi[si+1] = Utmpi[i+1];
        }
        /*No synchthreads needed in this case*/
        Dr[i] = (sUtmpr[si+1] - 2*sUtmpr[si] + sUtmpr[si-1])*lh2;
        Di[i] = (sUtmpi[si+1] - 2*sUtmpi[si] + sUtmpi[si-1])*lh2;
    }

    if(BC==2)   __syncthreads(); /* needed for MSD*/

        /*Boundery Conditions*/
    if(i == 0 || i == N-1){

        /*Boundary conditions:*/
        switch (BC){
            case 1: /*Dirichlet*/
                Dr[i] = -l_a*(s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - V[i])*sUtmpr[si];
                Di[i] = -l_a*(s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - V[i])*sUtmpi[si];
                break;
            case 2: /*MSD*/
                if(i==0)                msd_si = si+1;
                if(i==N-1)              msd_si = si-1;
                msd_i = i+(msd_si-si);

                Nb  = s*(sUtmpr[si]*sUtmpr[si]         + sUtmpi[si]*sUtmpi[si])         - V[i];
                Nb1 = s*(sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]) - V[msd_i];

                A   = (Dr[msd_i]*sUtmpr[msd_si]      + Di[msd_i]*sUtmpi[msd_si])/
                      (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);
                    
                Dr[i]  = (A + l_a*(Nb1-Nb))*sUtmpr[si];
                Di[i]  = (A + l_a*(Nb1-Nb))*sUtmpi[si];
                break;
            case 3: /*L0*/
                Dr[i]   = 0.0;
                Di[i]   = 0.0;
                break;
            case 4: /*1-sided*/
                if(i==0){
                   Dr[i] = (-Utmpr[3] + 4*Utmpr[2] - 5*sUtmpr[si+1] + 2*sUtmpr[si])*lh2;
                   Di[i] = (-Utmpi[3] + 4*Utmpi[2] - 5*sUtmpi[si+1] + 2*sUtmpi[si])*lh2;
                }
                else{
                   Dr[i] = (-Utmpr[N-4] + 4*Utmpr[N-3] - 5*sUtmpr[si-1] + 2*sUtmpr[si])*lh2;
                   Di[i] = (-Utmpi[N-4] + 4*Utmpi[N-3] - 5*sUtmpi[si-1] + 2*sUtmpi[si])*lh2;
                }
                break;
            default:
                Dr[i]   = 0.0;
                Di[i]   = 0.0;
                break;
       }/*BC Switch*/
    }/*BC*/
}/*computedx2*/

/*Kernel to evaluate F(Psi) and take RK4 sub-step*/
__global__ void compute_F_CD_F(float* ktotr, float* ktoti,
                               float* Utmpr, float* Utmpi,
                               float* Uoldr, float* Uoldi,
                               float* Uoutr, float* Uouti,
                               float* V, float s, float ah2,
                               int BC, int N, float k, int fstep)
{
    /*Declare shared memory space*/
    __shared__ float sUtmpr[BLOCK_SIZE+2];
    __shared__ float sUtmpi[BLOCK_SIZE+2];
    __shared__ float  NLSFr[BLOCK_SIZE+2];
    __shared__ float  NLSFi[BLOCK_SIZE+2];
    __shared__ float     sV[BLOCK_SIZE+2];

    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    int si = threadIdx.x + 1;
    int msd_si;
    float OM;
    /*Copy vector from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
        sV[si]     =     V[i];
    }

    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i > 0 && i < N-1)
    {
        /*Copy boundary layer of shared memory block*/
        if(si==1)
        {
            sUtmpr[si-1] = Utmpr[i-1];
            sUtmpi[si-1] = Utmpi[i-1];
        }
        if(si==blockDim.x)
        {
            sUtmpr[si+1] = Utmpr[i+1];
            sUtmpi[si+1] = Utmpi[i+1];
        }

        /*No synchthreads needed in this case*/
        NLSFr[si] = -ah2*(sUtmpi[si+1] - 2*sUtmpi[si] + sUtmpi[si-1])
                   -  (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
        NLSFi[si] =  ah2*(sUtmpr[si+1] - 2*sUtmpr[si] + sUtmpr[si-1])
                   +  (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
    }/*End of interier points*/

    if(BC==2)   __syncthreads(); /* needed for MSD*/

    /*Boundery Conditions*/
    if(i == 0 || i == N-1){
        switch (BC){
            case 1:
                NLSFr[si]   = 0.0f;
                NLSFi[si]   = 0.0f;
                break;
            case 2:
                if(i==0)     msd_si = si+1;
                if(i==N-1)   msd_si = si-1;

                OM = (NLSFi[msd_si]*sUtmpr[msd_si] - NLSFr[msd_si]*sUtmpi[msd_si])/
                    (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);

                NLSFr[si]  = -OM*sUtmpi[si];
                NLSFi[si]  =  OM*sUtmpr[si];
                break;
            case 3:
                NLSFr[si]   = - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si]   =   (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            case 4:
                if(i==0){
                NLSFr[si] = -ah2*(-Utmpi[3] + 4*Utmpi[2] - 5*sUtmpi[si+1] + 2*sUtmpi[si])
                            - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si] = ah2*(-Utmpr[3] + 4*Utmpr[2] - 5*sUtmpr[si+1] + 2*sUtmpr[si])
                            + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                }
                else{
                NLSFr[si] = -ah2*(-Utmpi[N-4] + 4*Utmpi[N-3] - 5*sUtmpi[si-1] + 2*sUtmpi[si])
                            - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si-1])*sUtmpi[si];
                NLSFi[si] =  ah2*(-Utmpr[N-4] + 4*Utmpr[N-3] - 5*sUtmpr[si-1] + 2*sUtmpr[si])
                            + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si-1])*sUtmpr[si];
                }
                break;
            default:
                NLSFr[si]   = 0.0f;
                NLSFi[si]   = 0.0f;
                break;
       }/*BC Switch*/
    }/*BC*/

    if(i<N){
      switch(fstep)  {
          case 1:
            ktotr[i] = NLSFr[si];
            ktoti[i] = NLSFi[si];
            /*sUtmp is really Uold and Uold is really Utmp*/
            Uoldr[i] = sUtmpr[si] + k*NLSFr[si];
            Uoldi[i] = sUtmpi[si] + k*NLSFi[si];
            break;
          case 2:
            ktotr[i] = ktotr[i] + 2*NLSFr[si];
            ktoti[i] = ktoti[i] + 2*NLSFi[si];
            Uoutr[i] = Uoldr[i] + k*NLSFr[si];
            Uouti[i] = Uoldi[i] + k*NLSFi[si];
            break;
          case 3:
            Uoldr[i] = Uoldr[i] + k*(ktotr[i] + NLSFr[si]);
            Uoldi[i] = Uoldi[i] + k*(ktoti[i] + NLSFi[si]);
            break;
        }/*switch step*/
    }/*i<N*/

}/*Compute_F_CD_F*/

/*Kernel to evaluate F(Psi) and take RK4 sub-step*/
__global__ void compute_F_CD(double* ktotr, double* ktoti,
                             double* Utmpr, double* Utmpi,
                             double* Uoldr, double* Uoldi,
                             double* Uoutr, double* Uouti,
                             double* V, double s, double ah2,
                             int BC, int N, double k, int fstep)
{
    /*Declare shared memory space*/
    __shared__ double sUtmpr[BLOCK_SIZE+2];
    __shared__ double sUtmpi[BLOCK_SIZE+2];
    __shared__ double  NLSFr[BLOCK_SIZE+2];
    __shared__ double  NLSFi[BLOCK_SIZE+2];
    __shared__ double     sV[BLOCK_SIZE+2];

    /*Create two indexes:  one for shared, one for global*/
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    int si = threadIdx.x + 1;
    int msd_si;
    double OM;
    /*Copy vector from global memory into shared memory*/
    if(i<N)
    {
        sUtmpr[si] = Utmpr[i];
        sUtmpi[si] = Utmpi[i];
        sV[si]     =     V[i];
    }

    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();

    /*If cell is not boundary...*/
    if (i > 0 && i < N-1)
    {
        /*Copy boundary layer of shared memory block*/
        if(si==1)
        {
            sUtmpr[si-1] = Utmpr[i-1];
            sUtmpi[si-1] = Utmpi[i-1];
        }
        if(si==blockDim.x)
        {
            sUtmpr[si+1] = Utmpr[i+1];
            sUtmpi[si+1] = Utmpi[i+1];
        }

        /*No synchthreads needed in this case*/
        NLSFr[si] = -ah2*(sUtmpi[si+1] - 2*sUtmpi[si] + sUtmpi[si-1])
                   -  (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
        NLSFi[si] =  ah2*(sUtmpr[si+1] - 2*sUtmpr[si] + sUtmpr[si-1])
                   +  (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
    }/*End of interier points*/

    if(BC==2)   __syncthreads(); /* needed for MSD*/

    /*Boundery Conditions*/
    if(i == 0 || i == N-1){
        switch (BC){
            case 1:
                NLSFr[si]   = 0.0;
                NLSFi[si]   = 0.0;
                break;
            case 2:
                if(i==0)     msd_si = si+1;
                if(i==N-1)   msd_si = si-1;

                OM = (NLSFi[msd_si]*sUtmpr[msd_si] - NLSFr[msd_si]*sUtmpi[msd_si])/
                    (sUtmpr[msd_si]*sUtmpr[msd_si] + sUtmpi[msd_si]*sUtmpi[msd_si]);

                NLSFr[si]  = -OM*sUtmpi[si];
                NLSFi[si]  =  OM*sUtmpr[si];
                break;
            case 3:
                NLSFr[si]   = - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si]   =   (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                break;
            case 4:
                if(i==0){
                NLSFr[si] = -ah2*(-Utmpi[3] + 4*Utmpi[2] - 5*sUtmpi[si+1] + 2*sUtmpi[si])
                            - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpi[si];
                NLSFi[si] = ah2*(-Utmpr[3] + 4*Utmpr[2] - 5*sUtmpr[si+1] + 2*sUtmpr[si])
                            + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si])*sUtmpr[si];
                }
                else{
                NLSFr[si] = -ah2*(-Utmpi[N-4] + 4*Utmpi[N-3] - 5*sUtmpi[si-1] + 2*sUtmpi[si])
                            - (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si-1])*sUtmpi[si];
                NLSFi[si] =  ah2*(-Utmpr[N-4] + 4*Utmpr[N-3] - 5*sUtmpr[si-1] + 2*sUtmpr[si])
                            + (s*(sUtmpr[si]*sUtmpr[si] + sUtmpi[si]*sUtmpi[si]) - sV[si-1])*sUtmpr[si];
                }
                break;
            default:
                NLSFr[si]   = 0.0;
                NLSFi[si]   = 0.0;
                break;
       }/*BC Switch*/
    }/*BC*/

    if(i<N){
      switch(fstep)  {
          case 1:
            ktotr[i] = NLSFr[si];
            ktoti[i] = NLSFi[si];
            /*sUtmp is really Uold and Uold is really Utmp*/
            Uoldr[i] = sUtmpr[si] + k*NLSFr[si];
            Uoldi[i] = sUtmpi[si] + k*NLSFi[si];
            break;
          case 2:
            ktotr[i] = ktotr[i] + 2*NLSFr[si];
            ktoti[i] = ktoti[i] + 2*NLSFi[si];
            Uoutr[i] = Uoldr[i] + k*NLSFr[si];
            Uouti[i] = Uoldi[i] + k*NLSFi[si];
            break;
          case 3:
            Uoldr[i] = Uoldr[i] + k*(ktotr[i] + NLSFr[si]);
            Uoldi[i] = Uoldi[i] + k*(ktoti[i] + NLSFi[si]);
            break;
        }/*switch step*/
    }/*i<N*/

}/*Compute_F_CD*/


extern "C" void NLSE1D_TAKE_STEPS_CUDA(double *Uoldr, double* Uoldi, double* V,
                                       double *Unewr, double* Unewi,
                                       double s, double a, double h2, int BC,
                                       int chunk_size, double k, int N, int method)
{
    double ah2,k2,k6,l_a,lh2,a76,a112;
    /*GPU variables:*/
    double *Uoutr, *Uouti, *ktotr, *ktoti;
    double *Utmpr, *Utmpi,*Dr, *Di;;
    double *Uoldr_gpu,*Uoldi_gpu, *V_gpu;
    int j;

    /*Precompute scalars:*/
    ah2 = a/h2;
    k2  = k/2.0;
    k6  = k/6.0;
    l_a  = 1.0/a;
    lh2  = 1.0/h2;
    a76  = a*(7.0/6.0);
    a112 = a*(1.0/12.0);    

    /*Allocate GPU arrays:*/
    hipMalloc( (void **) &Uoutr,    sizeof(double)*N);
    hipMalloc( (void **) &Uouti,    sizeof(double)*N);
    hipMalloc( (void **) &ktotr,    sizeof(double)*N);
    hipMalloc( (void **) &ktoti,    sizeof(double)*N);
    hipMalloc( (void **) &V_gpu,    sizeof(double)*N);
    hipMalloc( (void **) &Uoldr_gpu,sizeof(double)*N);
    hipMalloc( (void **) &Uoldi_gpu,sizeof(double)*N);
    hipMalloc( (void **) &Utmpr,    sizeof(double)*N);
    hipMalloc( (void **) &Utmpi,    sizeof(double)*N);
    if (method==2){
      hipMalloc( (void **) &Dr,       sizeof(double)*N);
      hipMalloc( (void **) &Di,       sizeof(double)*N);
    }

    /*Copy initial condition and potential vectors to GPU*/
    hipMemcpy( Uoldr_gpu, Uoldr, sizeof(double)*N,hipMemcpyHostToDevice);
    hipMemcpy( Uoldi_gpu, Uoldi, sizeof(double)*N,hipMemcpyHostToDevice);
    hipMemcpy( V_gpu,         V, sizeof(double)*N,hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((int)ceil((N+0.0)/dimBlock.x));

    /*Compute chunk of time steps using RK4*/
    if (method==1){
      for (j = 0; j<chunk_size; j++)
      {
        compute_F_CD<<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,V_gpu,V_gpu,s,ah2,BC,N,k2,1);
        compute_F_CD<<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,Uoutr,Uouti,V_gpu,s,ah2,BC,N,k2,2);
        compute_F_CD<<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoutr,Uouti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,s,ah2,BC,N,k, 2);
        compute_F_CD<<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,V_gpu,V_gpu,V_gpu,s,ah2,BC,N,k6,3);
      }
    }else if (method==2)
    {
      for (j = 0; j<chunk_size; j++)
      {
    	compute_D <<<dimGrid,dimBlock>>>(Dr,Di,Uoldr_gpu,Uoldi_gpu,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,V_gpu,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k2,1);
        compute_D <<<dimGrid,dimBlock>>>(Dr,Di,Utmpr,Utmpi,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,Uoutr,Uouti,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k2,2);
        compute_D <<<dimGrid,dimBlock>>>(Dr,Di,Uoutr,Uouti,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoutr,Uouti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k, 2);
        compute_D <<<dimGrid,dimBlock>>>(Dr,Di,Utmpr,Utmpi,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,V_gpu,V_gpu,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k6,3);
      }
    }

    /*Make sure everything is done*/
    hipDeviceSynchronize();

    /*Copy result from GPU back to CPU*/
    hipMemcpy(Unewr, Uoldr_gpu, sizeof(double)*N, hipMemcpyDeviceToHost);
    hipMemcpy(Unewi, Uoldi_gpu, sizeof(double)*N, hipMemcpyDeviceToHost);

    /*Free up GPU memory*/
    hipFree(Uoutr);
    hipFree(Uouti);
    hipFree(ktotr);
    hipFree(ktoti);
    hipFree(V_gpu);
    hipFree(Uoldr_gpu);
    hipFree(Uoldi_gpu);
    hipFree(Utmpr);
    hipFree(Utmpi);
    if (method==2){
      hipFree(Dr);
      hipFree(Di);
    }
}

extern "C" void NLSE1D_TAKE_STEPS_CUDA_F(float *Uoldr, float* Uoldi, float* V,
                                         float *Unewr, float* Unewi,
                                         float s, float a, float h2, int BC,
                                         int chunk_size, float k, int N, int method)
{
    float ah2,k2,k6,lh2,l_a,a76,a112;
    /*GPU variables:*/
    float *Uoutr, *Uouti, *ktotr, *ktoti;
    float *Utmpr, *Utmpi,*Dr, *Di;
    float *Uoldr_gpu,*Uoldi_gpu, *V_gpu;
    int j;

    /*Precompute scalars:*/
    ah2 = a/h2;
    k2  = k/2.0f;
    k6  = k/6.0f;
    l_a  = 1.0f/a;
    lh2  = 1.0f/h2;
    a76  = a*(7.0f/6.0f);
    a112 = a*(1.0f/12.0f);    

    /*Allocate GPU arrays:*/
    hipMalloc( (void **) &Uoutr,    sizeof(float)*N);
    hipMalloc( (void **) &Uouti,    sizeof(float)*N);
    hipMalloc( (void **) &ktotr,    sizeof(float)*N);
    hipMalloc( (void **) &ktoti,    sizeof(float)*N);
    hipMalloc( (void **) &V_gpu,    sizeof(float)*N);
    hipMalloc( (void **) &Uoldr_gpu,sizeof(float)*N);
    hipMalloc( (void **) &Uoldi_gpu,sizeof(float)*N);
    hipMalloc( (void **) &Utmpr,    sizeof(float)*N);
    hipMalloc( (void **) &Utmpi,    sizeof(float)*N);
    if (method==2){
      hipMalloc( (void **) &Dr,     sizeof(float)*N);
      hipMalloc( (void **) &Di,     sizeof(float)*N);
    }    

    /*Copy initial condition and potential vectors to GPU*/
    hipMemcpy( Uoldr_gpu, Uoldr, sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy( Uoldi_gpu, Uoldi, sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy( V_gpu,         V, sizeof(float)*N,hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((int)ceil((N+0.0)/dimBlock.x));

    /*Compute chunk of time steps using RK4*/
    if (method==1){      
      for (j = 0; j<chunk_size; j++)
      {
        compute_F_CD_F<<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,V_gpu,V_gpu,s,ah2,BC,N,k2,1);
        compute_F_CD_F<<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,Uoutr,Uouti,V_gpu,s,ah2,BC,N,k2,2);
        compute_F_CD_F<<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoutr,Uouti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,s,ah2,BC,N,k, 2);
        compute_F_CD_F<<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,V_gpu,V_gpu,V_gpu,s,ah2,BC,N,k6,3);
      }
    }else if (method==2){   
      for (j = 0; j<chunk_size; j++)
      {
        compute_D_F <<<dimGrid,dimBlock>>>(Dr,Di,Uoldr_gpu,Uoldi_gpu,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,V_gpu,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k2,1);
        compute_D_F <<<dimGrid,dimBlock>>>(Dr,Di,Utmpr,Utmpi,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,Uoutr,Uouti,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k2,2);
        compute_D_F <<<dimGrid,dimBlock>>>(Dr,Di,Uoutr,Uouti,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoutr,Uouti,Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k, 2);
        compute_D_F <<<dimGrid,dimBlock>>>(Dr,Di,Utmpr,Utmpi,V_gpu,lh2,l_a,s,BC,N);
        compute_F_2SHOC_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,Utmpi,Uoldr_gpu,Uoldi_gpu,V_gpu,V_gpu,Dr,Di,V_gpu,s,a,a76,a112,BC,N,k6,3);
      }	    
    }

    /*Make sure everything is done*/
    hipDeviceSynchronize();
	
    /*Copy result from GPU back to CPU*/
    hipMemcpy(Unewr, Uoldr_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(Unewi, Uoldi_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);

    /*Free up GPU memory*/
    hipFree(Uoutr);
    hipFree(Uouti);
    hipFree(ktotr);
    hipFree(ktoti);
    hipFree(V_gpu);
    hipFree(Uoldr_gpu);
    hipFree(Uoldi_gpu);
    hipFree(Utmpr);
    hipFree(Utmpi);
    if (method==2){
      hipFree(Dr);
      hipFree(Di);
    }    
	
	hipDeviceReset();
	
}
