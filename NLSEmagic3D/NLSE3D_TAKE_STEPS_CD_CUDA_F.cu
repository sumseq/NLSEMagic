#include "hip/hip_runtime.h"
/*----------------------------
NLSE3D_TAKE_STEPS_CD_CUDA_F.cu:
Program to integrate a chunk of time steps of the 2D Nonlinear Shrodinger Equation
i*Ut + a*(Uxx + Uyy + Uzz) - V(r)U + s*|U|^2*U = 0
using RK4 + CD with CUDA compatable GPUs in
single precision.

Ronald N Caplan
Computational Science Research Center
San Diego State University

INPUT:
(U,V,s,a,h2,BC,chunk_size,k)
U  = Current solution matrix
V  = External Potential matrix
s  = Nonlinearity paramater
a  = Laplacian paramater
h2 = Spatial step size squared (h^2)
BC = Boundary condition selection switch:  1: Dirchilet 2:MSD 3:Lap=0
chunk_size = Number of time steps to take
k  = Time step size

OUTPUT:
U:  New solution matrix
-------------------------------*/

#include "hip/hip_runtime.h"
#include "mex.h"
#include "math.h"

/*Define block size*/
const int BLOCK_SIZEX = 16;
const int BLOCK_SIZEY = 16;
const int BLOCK_SIZEZ = 4;

/*Kernel to evaluate F(Psi) using shared memory*/
__global__ void compute_F(float* ktotr, float* ktoti,
                          float* Utmpr, float* Utmpi,
                          float* Uoldr, float* Uoldi,
                          float* Uoutr, float* Uouti,
                          float* V, float s, float ah2,
                          int BC, int L, int N, int M, int gridDim_y, float K, int fstep)
{
    /*Declare shared memory space*/
    __shared__ float sUtmpr[BLOCK_SIZEZ+2][BLOCK_SIZEY+2][BLOCK_SIZEX+2];
    __shared__ float sUtmpi[BLOCK_SIZEZ+2][BLOCK_SIZEY+2][BLOCK_SIZEX+2];
    __shared__ float  NLSFr[BLOCK_SIZEZ+2][BLOCK_SIZEY+2][BLOCK_SIZEX+2];
    __shared__ float  NLSFi[BLOCK_SIZEZ+2][BLOCK_SIZEY+2][BLOCK_SIZEX+2];
    __shared__ float     sV[BLOCK_SIZEZ+2][BLOCK_SIZEY+2][BLOCK_SIZEX+2];

    /*Create six indexes:  three for shared, three for global*/
    int i, j, k, blockIdx_z, blockIdx_y;
    /*Compute idx for z in cube (int division acts as floor operator here)*/
    blockIdx_z  = blockIdx.y/gridDim_y;
    /*Compute "true" idx for y in cube*/
    blockIdx_y  = blockIdx.y - blockIdx_z*gridDim_y;
    /*Now can compute j and k as if there was a 3D CUDA grid:*/
    k = blockIdx.x*blockDim.x + threadIdx.x;
    j = blockIdx_y*blockDim.y + threadIdx.y;
    i = blockIdx_z*blockDim.z + threadIdx.z;

    int sk  = threadIdx.x + 1;
    int sj  = threadIdx.y + 1;
    int si  = threadIdx.z + 1;

    int msd_si, msd_sj, msd_sk;
    float OM;

    int ijk = N*M*i + M*j + k;

    if(i<L && (j<N && k<M)){
        /*Copy blocksized matrix from global memory into shared memory*/
        sUtmpr[si][sj][sk] = Utmpr[ijk];
        sUtmpi[si][sj][sk] = Utmpi[ijk];
            sV[si][sj][sk] =     V[ijk];
    }


    /*Synchronize the threads in the block so that all shared cells are filled.*/
    __syncthreads();


    /*If cell is NOT on boundary...*/
    if ( (j>0 && j<N-1) && ((i>0 && i<L-1) && (k>0 && k<M-1)) )
    {
         /*Copy boundary layer of shared memory block*/
        if(si==1)
        {
            sUtmpr[0][sj][sk] = Utmpr[ijk - N*M];
            sUtmpi[0][sj][sk] = Utmpi[ijk - N*M];
        }
        if(si==blockDim.z)
        {
            sUtmpr[si+1][sj][sk] = Utmpr[ijk + N*M];
            sUtmpi[si+1][sj][sk] = Utmpi[ijk + N*M];
        }
        if(sj==1)
        {
            sUtmpr[si][0][sk] = Utmpr[ijk - M];
            sUtmpi[si][0][sk] = Utmpi[ijk - M];
        }
        if(sj==blockDim.y)
        {
            sUtmpr[si][sj+1][sk] = Utmpr[ijk + M];
            sUtmpi[si][sj+1][sk] = Utmpi[ijk + M];
        }
        if(sk==1)
        {
            sUtmpr[si][sj][0] = Utmpr[ijk - 1];
            sUtmpi[si][sj][0] = Utmpi[ijk - 1];
        }
        if(sk==blockDim.x)
        {
            sUtmpr[si][sj][sk+1] = Utmpr[ijk + 1];
            sUtmpi[si][sj][sk+1] = Utmpi[ijk + 1];
        }

        /*No synchthreads needed in this case*/

        NLSFr[si][sj][sk] = -ah2*(sUtmpi[si+1][sj][sk] - 6*sUtmpi[si][sj][sk] + sUtmpi[si-1][sj][sk] +
                           sUtmpi[si][sj+1][sk]                        + sUtmpi[si][sj-1][sk] +
                           sUtmpi[si][sj][sk+1]                        + sUtmpi[si][sj][sk-1])
                         + (sV[si][sj][sk] - s*(sUtmpr[si][sj][sk]*sUtmpr[si][sj][sk] +
                                        sUtmpi[si][sj][sk]*sUtmpi[si][sj][sk]))*sUtmpi[si][sj][sk];

        NLSFi[si][sj][sk] =  ah2*(sUtmpr[si+1][sj][sk] - 6*sUtmpr[si][sj][sk] + sUtmpr[si-1][sj][sk] +
                           sUtmpr[si][sj+1][sk]                        + sUtmpr[si][sj-1][sk] +
                           sUtmpr[si][sj][sk+1]                        + sUtmpr[si][sj][sk-1])
                         - (sV[si][sj][sk] - s*(sUtmpr[si][sj][sk]*sUtmpr[si][sj][sk] +
                                       sUtmpi[si][sj][sk]*sUtmpi[si][sj][sk]))*sUtmpr[si][sj][sk];

    }/*End of interier points*/

    /*This synch is needed for the MSD boundary condition*/
    if(BC==2)  __syncthreads();

    if(i<L && (j<N && k<M)){

        /*Cell is ON Boundery*/
        if(!( (j>0 && j<N-1) && ((i>0 && i<L-1) && (k>0 && k<M-1)) ) ){
            switch(BC){
                case 1: /*Dirichlet*/
                    NLSFr[si][sj][sk]   = 0.0f;
                    NLSFi[si][sj][sk]   = 0.0f;
                    break;
                case 2: /* Mod-Squared Dirichlet |U|^2=B */
                    if(i==0)                msd_si = si+1;
                    if(i==L-1)              msd_si = si-1;
                    if((i!=0) && (i!=L-1))  msd_si = si;
                    if(j==0)                msd_sj = sj+1;
                    if(j==N-1)              msd_sj = sj-1;
                    if((j!=0) && (j!=N-1))  msd_sj = sj;
                    if(k==0)                msd_sk = sk+1;
                    if(k==M-1)              msd_sk = sk-1;
                    if((k!=0) && (k!=M-1))  msd_sk = sk;

                    if(sUtmpr[msd_si][msd_sj][msd_sk]==0 && sUtmpi[msd_si][msd_sj][msd_sk]==0)
                    {
                        OM=0;
                    }
                    else{ 
                        OM = (NLSFi[msd_si][msd_sj][msd_sk]*sUtmpr[msd_si][msd_sj][msd_sk] -  NLSFr[msd_si][msd_sj][msd_sk]*sUtmpi[msd_si][msd_sj][msd_sk])/
                         (sUtmpr[msd_si][msd_sj][msd_sk]*sUtmpr[msd_si][msd_sj][msd_sk] + sUtmpi[msd_si][msd_sj][msd_sk]*sUtmpi[msd_si][msd_sj][msd_sk]);
                    }                       
                    NLSFr[si][sj][sk]  = -OM*sUtmpi[si][sj][sk];
                    NLSFi[si][sj][sk]  =  OM*sUtmpr[si][sj][sk];
                    break;
                case 3: /*Uxx+Uyy+Uzz=0:*/
                    NLSFr[si][sj][sk] = - (s*(sUtmpr[si][sj][sk]*sUtmpr[si][sj][sk] + sUtmpi[si][sj][sk]*sUtmpi[si][sj][sk]) - sV[si][sj][sk])*sUtmpi[si][sj][sk];
                    NLSFi[si][sj][sk] =   (s*(sUtmpr[si][sj][sk]*sUtmpr[si][sj][sk] + sUtmpi[si][sj][sk]*sUtmpi[si][sj][sk]) - sV[si][sj][sk])*sUtmpr[si][sj][sk];
                    break;
                default:
                    NLSFr[si][sj][sk]   = 0.0f;
                    NLSFi[si][sj][sk]   = 0.0f;
                    break;
            }/*BC switch*/
        }/*on BC*/

        switch(fstep)  {
          case 1:
            ktotr[ijk] = NLSFr[si][sj][sk];
            ktoti[ijk] = NLSFi[si][sj][sk];
            /*sUtmp is really Uold and Uold is really Utmp*/
            Uoldr[ijk] = sUtmpr[si][sj][sk] + K*NLSFr[si][sj][sk];
            Uoldi[ijk] = sUtmpi[si][sj][sk] + K*NLSFi[si][sj][sk];
            break;
          case 2:
            ktotr[ijk] = ktotr[ijk] + 2*NLSFr[si][sj][sk];
            ktoti[ijk] = ktoti[ijk] + 2*NLSFi[si][sj][sk];
            Uoutr[ijk] = Uoldr[ijk] + K*NLSFr[si][sj][sk];
            Uouti[ijk] = Uoldi[ijk] + K*NLSFi[si][sj][sk];
            break;
          case 3:
            Uoldr[ijk] = Uoldr[ijk] + K*(ktotr[ijk] + NLSFr[si][sj][sk]);
            Uoldi[ijk] = Uoldi[ijk] + K*(ktoti[ijk] + NLSFi[si][sj][sk]);
            break;

        }/*switch step*/
    }/*<end*/
}/*Compute_F*/

/*Main mex function*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    int    L,N,M,dims,gridDim_y;
    const int *dim_array;
    int    i, c, chunk_size, BC;
    float  h2, a, ah2, s, K, K2, K6;
    double *vUoldr, *vUoldi,  *vV, *vUnewr, *vUnewi;
    float  *fvUoldr, *fvUoldi,  *fvV;

    /*GPU variables:*/
    float *Utmpr, *Utmpi;
    float *Uoutr, *Uouti, *ktotr, *ktoti;
    float *Uoldr_gpu, *Uoldi_gpu,  *V_gpu;

    /*Find the dimensions of the input cube*/
    dims      = (int)mxGetNumberOfDimensions(prhs[0]);
    dim_array = (const int*)mxGetDimensions(prhs[0]);
    M         = dim_array[0];
    N         = dim_array[1];
    L         = dim_array[2];

    /*Create output vector*/
    plhs[0] =  mxCreateNumericArray((mwSize)dims, (mwSize*)dim_array, mxDOUBLE_CLASS, mxCOMPLEX);

    /* Retrieve the input data */
    vUoldr = mxGetPr(prhs[0]);
    if(mxIsComplex(prhs[0])){
        vUoldi = mxGetPi(prhs[0]);
    }
    else{
        vUoldi = (double *)malloc(sizeof(double)*L*N*M);
        for(i=0;i<L*N*M;i++){
            vUoldi[i] = 0.0;
        }
    }
    vV     = mxGetPr(prhs[1]);

    /*Get the rest of the input variables*/
    s           = (float)mxGetScalar(prhs[2]);
    a           = (float)mxGetScalar(prhs[3]);
    h2          = (float)mxGetScalar(prhs[4]);
    BC          =   (int)mxGetScalar(prhs[5]);
    chunk_size  =   (int)mxGetScalar(prhs[6]);
    K           = (float)mxGetScalar(prhs[7]);

    /*Pre-compute parameter divisions*/
    ah2 = a/h2;
    K2  = K/2.0f;
    K6  = K/6.0f;

    /*Allocate float input vectors*/
    fvV     = (float*)malloc(sizeof(float)*L*N*M);
    fvUoldr = (float*)malloc(sizeof(float)*L*N*M);
    fvUoldi = (float*)malloc(sizeof(float)*L*N*M);

    /*Allocate 1D CUDA memory*/
    hipMalloc((void**) &Uoldr_gpu, M*N*L*sizeof(float));
    hipMalloc((void**) &Uoldi_gpu, M*N*L*sizeof(float));
    hipMalloc((void**) &V_gpu,     M*N*L*sizeof(float));
    hipMalloc((void**) &Utmpr,     M*N*L*sizeof(float));
    hipMalloc((void**) &Utmpi,     M*N*L*sizeof(float));
    hipMalloc((void**) &Uouti,     M*N*L*sizeof(float));
    hipMalloc((void**) &Uoutr,     M*N*L*sizeof(float));
    hipMalloc((void**) &ktotr,     M*N*L*sizeof(float));
    hipMalloc((void**) &ktoti,     M*N*L*sizeof(float));

        /*Convert double input vectors to float*/
    for(i=0;i<L*N*M;i++){
      fvV[i]     = (float)vV[i];
      fvUoldr[i] = (float)vUoldr[i];
      fvUoldi[i] = (float)vUoldi[i];
    }

    /*Copy input vectors to GPU*/
    hipMemcpy(Uoldr_gpu, fvUoldr, M*N*L*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Uoldi_gpu, fvUoldi, M*N*L*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(V_gpu,     fvV,     M*N*L*sizeof(float),hipMemcpyHostToDevice);

    /*Set up CUDA grid and block size*/
    dim3 dimBlock(BLOCK_SIZEX,BLOCK_SIZEY,BLOCK_SIZEZ);

    /*Compute desired y grid dimension*/
    gridDim_y  = (int)ceil((N+0.0f)/dimBlock.y);

    /*For 3D need to extend y-grid dimention to include z-cuts:*/
    dim3 dimGrid((int)ceil((M+0.0)/dimBlock.x), gridDim_y*((int)ceil((L+0.0)/dimBlock.z)));

    /*Compute chunk of time steps*/
    for (c=0; c<chunk_size; c++)
    {
      compute_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoldr_gpu,Uoldi_gpu,Utmpr,    Utmpi,    V_gpu,V_gpu,V_gpu,s,ah2,BC,L,N,M,gridDim_y,K2,1);
      compute_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,    Utmpi,    Uoldr_gpu,Uoldi_gpu,Uoutr,Uouti,V_gpu,s,ah2,BC,L,N,M,gridDim_y,K2,2);
      compute_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Uoutr,    Uouti,    Uoldr_gpu,Uoldi_gpu,Utmpr,Utmpi,V_gpu,s,ah2,BC,L,N,M,gridDim_y,K, 2);
      compute_F <<<dimGrid,dimBlock>>>(ktotr,ktoti,Utmpr,    Utmpi,    Uoldr_gpu,Uoldi_gpu,V_gpu,V_gpu,V_gpu,s,ah2,BC,L,N,M,gridDim_y,K6,3);
    }

    /*Set up output vectors*/
    vUnewr = mxGetPr(plhs[0]);
    vUnewi = mxGetPi(plhs[0]);

    /*Make sure everything is done (important for large chunk-size computations)*/
    hipDeviceSynchronize();

    /*Transfer solution back to CPU*/
    hipMemcpy(fvUoldr,Uoldr_gpu, M*N*L*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(fvUoldi,Uoldi_gpu, M*N*L*sizeof(float),hipMemcpyDeviceToHost);

    /*Convert float vector to double output vector*/
    for(i=0;i<L*N*M;i++){
        vUnewr[i] = (double)fvUoldr[i];
        vUnewi[i] = (double)fvUoldi[i];
    }

    /*Free up GPU memory*/
    hipFree(Uoutr);
    hipFree(Uouti);
    hipFree(ktotr);
    hipFree(ktoti);
    hipFree(V_gpu);
    hipFree(Uoldr_gpu);
    hipFree(Uoldi_gpu);
    hipFree(Utmpr);
    hipFree(Utmpi);

    /*Free up CPU memory*/
    free(fvUoldr);
    free(fvUoldi);
    free(fvV);

    if(!mxIsComplex(prhs[0])){
        free(vUoldi);
    }

    hipDeviceReset();

}

/*For reference, command to compile code in MATLAB on windows:
nvmex -f nvmexopts.bat NLSE3D_TAKE_STEPS_CD_CUDA_F.cu -IC:\cuda\include -LC:\cuda\lib -lcudart
*/
